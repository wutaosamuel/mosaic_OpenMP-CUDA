#include "hip/hip_runtime.h"
//#ifndef __HIPCC__
//#define __HIPCC__
//#endif

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include ""

#define USER_NAME "acp18tw"		//replace with your user name

typedef enum MODE { CPU, OPENMP, CUDA, ALL } MODE;
typedef enum PPM_FORMAT { PPM_BINARY, PPM_PLAIN_TEXT } PPM_FORMAT;

#define FAILURE 0
#define SUCCESS !FAILURE
#define THREADS_PRE_BLOCK_X_32 32
#define THREADS_PRE_BLOCK_Y_32 32

struct PPMrgb{
  unsigned char red;
  unsigned char green;
  unsigned char blue;
};

unsigned int c = 0;

struct PPMrgb* readPPMFile(char *filename, unsigned int *w, unsigned int *h, unsigned int *d);
int writePPMFile(char *filename, struct PPMrgb *rgb, unsigned int *w, unsigned int *h, unsigned int *d, unsigned int m);
void freeAll(struct PPMrgb *rgb, unsigned int *w, unsigned int *h, unsigned int *d);
void checkCUDAError(const char *msg);
void print_help();

__global__ void mosaicCal_cuda(float *inValue, float *ouValue, float *w, float *h, float *mSize)
{
  // share memory of image
  extern __shared__ float buffer[];
  // share memory of mosaic size in x and y direction
  __shared__ float blockSize_x;
  __shared__ float blockSize_y;
  __shared__ int width_int, height_int, mosaic_size;
  width_int = (int) (*w);
  height_int = (int) (*h);
  mosaic_size = (int) (*mSize);

  int i,j;
  int rgbIndex;
  // calculate position index in block;
  int index = blockDim.x * threadIdx.y + threadIdx.x;
  int w_index = blockIdx.x * mosaic_size + threadIdx.x;
  int h_index = blockIdx.y * mosaic_size + threadIdx .y;

  // initialize burffer
  buffer[index] = 0;
  if (index == 0) {
    blockSize_x = 0.0;
    blockSize_y = 0.0;
  }

  // calculate number pixels in a filter/block
  if (blockIdx.x == (gridDim.x - 1) && width_int % mosaic_size != 0)
    blockSize_x = (float)(width_int % mosaic_size);
  else
    blockSize_x = mosaic_size;
  if (blockIdx.y == (gridDim.y - 1) && height_int % mosaic_size != 0)
    blockSize_y = (float)(height_int % mosaic_size);
  else
    blockSize_y = mosaic_size;

  __syncthreads();

  // allocate the pixels to buffer
  //rgbIndex = h_index * width_int + w_index;
  for (j = threadIdx.y; j < blockSize_y; j += blockDim.y) {
    for(i = threadIdx.x; i < blockSize_x; i += blockDim.x) {
      rgbIndex = (blockIdx.y * mosaic_size + j) * width_int + w_index;
      buffer[index] += inValue[rgbIndex];
      rgbIndex += blockDim.x;
    }
    //rgbIndex = (blockIdx.y * mosaic_size + j) * width_int + w_index;
  }
  __syncthreads();

  // get total sum in buffer[0]
  for (unsigned int stride = blockDim.x*blockDim.y/2; stride > 0; stride >>= 1) {
    if (index < stride){
      buffer[index] += buffer[index + stride];
    }
    __syncthreads();
  }
  __syncthreads();

  // average the sum at buffer[0]
  if (index == 0) {
    buffer[0] = buffer[0] / (blockSize_x * blockSize_y);
  }
  __syncthreads();

  // resize to original size
  for (j = threadIdx.y; j < blockSize_y; j += blockDim.y) {
    for(i = threadIdx.x; i < blockSize_x; i += blockDim.x) {
      rgbIndex = (blockIdx.y * mosaic_size + j) * width_int + w_index;
      ouValue[rgbIndex] = buffer[0];
      rgbIndex += blockDim.x;
    }
  }
}

__global__ void average_cuda(float *inpt, float *oupt, float *w, float *h)
{
  extern __shared__ float tmp[];
  unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  // insert to shared memory
  tmp[threadIdx.x] = inpt[idx];

  __syncthreads();

  // calculate average
  for (unsigned int stride = blockDim.x/2; stride > 0; stride >>= 1) {
    if (threadIdx.x < stride){
      tmp[threadIdx.x] += tmp[threadIdx.x + stride];
    }
    __syncthreads();
  }

  // calculate block sum, add array[0] value to output
  if (threadIdx.x == 0) {
    atomicAdd(oupt, tmp[0]);
  }
}

int main() {
  char input_file[] = "Sheffield16x16.ppm";
  //char input_file[] = "SheffieldPlainText16x16.ppm";
  //char input_file[] = "Sheffield512x512.ppm";
  //char input_file[] = "DogPlainText2048x2048.ppm";
  char output_file[] = "testText.ppm";
  struct PPMrgb *ppmrgb_input;
  struct PPMrgb *cudargb;
  unsigned int *width_, *height_, *deepth_;
  unsigned int magic_number;
  float *r_float, *g_float, *b_float;
  float *r_output, *g_output, *b_output;
  float *r_sum, *g_sum, *b_sum;
  float width_float, height_float;
  c = 4;
  float c_float = (float)c;
  float msec;
  hipEvent_t cuda_start, cuda_stop;

  printf("Start mosaic calculation in cuda\n");
  // host memory
  width_  = (unsigned int*)malloc(sizeof(unsigned int));
  height_ = (unsigned int*)malloc(sizeof(unsigned int));
  deepth_ = (unsigned int*)malloc(sizeof(unsigned int));
  r_sum = (float *)malloc(sizeof(float));
  g_sum = (float *)malloc(sizeof(float));
  b_sum = (float *)malloc(sizeof(float));

  // host memory 2, after read file
  //ppmrgb_input = readPPMFile(input_fileName, width_, height_, deepth_);
  ppmrgb_input = readPPMFile(input_file, width_, height_, deepth_);
  cudargb = (struct PPMrgb*)malloc(sizeof(struct PPMrgb)* (*width_) * (*height_));
  r_float = (float *)malloc(sizeof(float)*(*width_)*(*height_));
  g_float = (float *)malloc(sizeof(float)*(*width_)*(*height_));
  b_float = (float *)malloc(sizeof(float)*(*width_)*(*height_));
  r_output = (float *)malloc(sizeof(float)*(*width_)*(*height_));
  g_output = (float *)malloc(sizeof(float)*(*width_)*(*height_));
  b_output = (float *)malloc(sizeof(float)*(*width_)*(*height_));
  for(int i=0; i<(*width_)*(*height_); i++){
    r_float[i] = (float)ppmrgb_input[i].red;
    g_float[i] = (float)ppmrgb_input[i].green;
    b_float[i] = (float)ppmrgb_input[i].blue;
  }

  //printf("%d %d %d\n", *width_, *height_, *deepth_);
  width_float = (float)(*width_);
  height_float = (float)(*height_);

  // prepare for device memory
  float *d_width_float, *d_height_float, *d_c_float;
  float *d_r_float, *d_g_float, *d_b_float;
  float *d_r_output, *d_g_output, *d_b_output;
  float *d_r_sum, *d_g_sum, *d_b_sum;
  // set pointer to float type variables
  float *p_c_float = &c_float;
  float *p_width_float = &width_float;
  float *p_height_float = &height_float;

  // allocate device memory
  hipMalloc((void **)&d_c_float, sizeof(float));
  hipMalloc((void **)&d_width_float, sizeof(float));
  hipMalloc((void **)&d_height_float, sizeof(float));
  hipMalloc((void **)&d_r_float, sizeof(float)*(*width_)*(*height_));
  hipMalloc((void **)&d_g_float, sizeof(float)*(*width_)*(*height_));
  hipMalloc((void **)&d_b_float, sizeof(float)*(*width_)*(*height_));
  checkCUDAError("CUDA malloc for must use");

  // copy necessary value to device
  hipMemcpy(d_c_float, p_c_float, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_width_float, p_width_float, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_height_float, p_height_float, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_r_float, r_float, sizeof(float)*(*width_)*(*height_), hipMemcpyHostToDevice);
  hipMemcpy(d_g_float, g_float, sizeof(float)*(*width_)*(*height_), hipMemcpyHostToDevice);
  hipMemcpy(d_b_float, b_float, sizeof(float)*(*width_)*(*height_), hipMemcpyHostToDevice);
  checkCUDAError("CUDA memcpy for must use");

  // calculate average rgb value
  hipMalloc((void **)&d_r_sum, sizeof(float));
  hipMalloc((void **)&d_g_sum, sizeof(float));
  hipMalloc((void **)&d_b_sum, sizeof(float));
  checkCUDAError("CUDA malloc for output sum");
  // start calculate seperated by RGB value
  int threads_num = 32;
  int blocks_num = (*width_)*(*height_) / threads_num;
  average_cuda<<<blocks_num, threads_num, sizeof(float)*32>>>(d_r_float, d_r_sum, d_width_float, d_height_float);
  checkCUDAError("average for red");
  average_cuda<<<blocks_num, threads_num, sizeof(float)*32>>>(d_g_float, d_g_sum, d_width_float, d_height_float);
  checkCUDAError("average for green");
  average_cuda<<<blocks_num, threads_num, sizeof(float)*32>>>(d_b_float, d_b_sum, d_width_float, d_height_float);
  checkCUDAError("average for blue");
  // copy value from device to host
  hipMemcpy(r_sum, d_r_sum, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(g_sum, d_g_sum, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(b_sum, d_b_sum, sizeof(float), hipMemcpyDeviceToHost);
  checkCUDAError("CUDA memcpy for output");
  printf("The cuda red = %.1f, green = %.1f, blue = %.1f\n", (*r_sum)/(width_float*height_float), (*g_sum)/(width_float*height_float), (*b_sum)/(width_float*height_float));
  // free average calculation memory
  hipFree(d_r_sum);hipFree(d_g_sum);hipFree(d_b_sum);
  free(r_sum);free(g_sum);free(b_sum);

  // start to mosaic calculation
  // set block size and grid size
  dim3 threads_pre_block, blocks_pre_grid;
  if ( 0 < c && c <= 8) {
    threads_pre_block.x = 8;
    threads_pre_block.y = 8;
  }else if (8 < c && c <= 16) {
    threads_pre_block.x = 16;
    threads_pre_block.y = 16;
  }else if (16 < c && c <= 32) {
    threads_pre_block.x = THREADS_PRE_BLOCK_X_32;
    threads_pre_block.y = THREADS_PRE_BLOCK_Y_32;
  }else{
		fprintf(stderr, "Error: CUDA Implementation not support for over 32X32 mosaic size.\n");
		print_help();
    return 1;
  }
  blocks_pre_grid.x = ((*width_) + c - 1) / c;
  blocks_pre_grid.y = ((*height_) + c - 1) / c;
  // allocate shared memory size
  int memsize = sizeof(float) * (threads_pre_block.x * threads_pre_block.y);

  // allocate device memory
  hipMalloc((void **)&d_r_output, sizeof(float)*(*width_)*(*height_));
  hipMalloc((void **)&d_g_output, sizeof(float)*(*width_)*(*height_));
  hipMalloc((void **)&d_b_output, sizeof(float)*(*width_)*(*height_));
  checkCUDAError("CUDA malloc for mosaic output");
  // Allocate cuda events for timing
  hipEventCreate(&cuda_start);
  hipEventCreate(&cuda_stop);
  checkCUDAError("CUDA event creation");

  // count time and mosaic calculation in RGB value
  hipEventRecord(cuda_start);
  mosaicCal_cuda<<<blocks_pre_grid, threads_pre_block, memsize>>>(d_r_float, d_r_output, d_width_float, d_height_float, d_c_float);
  checkCUDAError("mosaic red");
  mosaicCal_cuda<<<blocks_pre_grid, threads_pre_block, memsize>>>(d_g_float, d_g_output, d_width_float, d_height_float, d_c_float);
  checkCUDAError("mosaic green");
  mosaicCal_cuda<<<blocks_pre_grid, threads_pre_block, memsize>>>(d_b_float, d_b_output, d_width_float, d_height_float, d_c_float);
  checkCUDAError("mosaic blue");
  hipEventRecord(cuda_stop);
  hipEventSynchronize(cuda_stop);
  checkCUDAError("CUDA kernel execution and timing");
  hipEventElapsedTime(&msec, cuda_start, cuda_stop);
  hipDeviceSynchronize();
  checkCUDAError("CUDA timing");

  // copy device memory to host
  hipMemcpy(r_output, d_r_output, sizeof(float)*(*width_)*(*height_), hipMemcpyDeviceToHost);
  hipMemcpy(g_output, d_g_output, sizeof(float)*(*width_)*(*height_), hipMemcpyDeviceToHost);
  hipMemcpy(b_output, d_b_output, sizeof(float)*(*width_)*(*height_), hipMemcpyDeviceToHost);
  checkCUDAError("CUDA memcpy for output to host");

  // write back to PPM size
  for(int i=0; i<(*width_)*(*height_); i++){
    cudargb[i].red = (unsigned char)r_output[i];
    cudargb[i].green = (unsigned char)g_output[i];
    cudargb[i].blue = (unsigned char)b_output[i];
  }
  printf("The cuda time is: %.2fms\n", msec);
  //if (writePPMFile(output_fileName, cudargb, width_, height_, deepth_, ppm_format) == FAILURE)
  if (writePPMFile(output_file, cudargb, width_, height_, deepth_, 3) == FAILURE)
    return 1;

  // free all
  hipFree(d_width_float);hipFree(d_height_float);
  hipFree(d_c_float);hipFree(d_r_float);hipFree(d_g_float);hipFree(d_b_float);
  hipFree(d_r_output);hipFree(d_g_output);hipFree(d_b_output);
  freeAll(ppmrgb_input, width_, height_, deepth_);
  printf("done\n");

}

void print_help(){
	printf("mosaic_%s C M -i input_file -o output_file [options]\n", USER_NAME);

	printf("where:\n");
	printf("\tC              Is the mosaic cell size which should be any positive\n"
		   "\t               power of 2 number \n");
	printf("\tM              Is the mode with a value of either CPU, OPENMP, CUDA or\n"
		   "\t               ALL. The mode specifies which version of the simulation\n"
		   "\t               code should execute. ALL should execute each mode in\n"
		   "\t               turn.\n");
	printf("\t-i input_file  Specifies an input image file\n");
	printf("\t-o output_file Specifies an output image file which will be used\n"
		   "\t               to write the mosaic image\n");
	printf("[options]:\n");
	printf("\t-f ppm_format  PPM image output format either PPM_BINARY (default) or \n"
		   "\t               PPM_PLAIN_TEXT\n ");
}

struct PPMrgb* readPPMFile(char *filename, unsigned int *w, unsigned int *h, unsigned int *d)
{
  int line = 0;
  unsigned int getW, getH, getD;
  int i,j;
  char header[4][100];
  struct PPMrgb *rgb;
  FILE *f = NULL;
  f = fopen(filename, "r");

  // get first 4 parameters
  while(line < 4){
    fgets(header[line], 100, f);
    if (strncmp(header[line], "#", 1) != 0)
      ++line;
  }

  // get or check first 4 parameters
  getW = (unsigned int)strtol(header[1], NULL, 10);
  getH = (unsigned int)strtol(header[2], NULL, 10);
  getD = (unsigned int)strtol(header[3], NULL, 10);
  *w = getW;
  *h = getH;
  *d = getD;
  rgb = (struct PPMrgb*)malloc(sizeof(struct PPMrgb)* (*w) * (*h));
  if (strncmp(header[0], "P3", 2) == 0) {
    for (i = 0; i < *h; i++){
      for (j = 0; j < *w; j++) {
        if (j == (*w) -1)
          fscanf(f, "%hhu %hhu %hhu\n", &rgb[i*(*w)+j].red, &rgb[i*(*w)+j].green, &rgb[i*(*w)+j].blue);
        else
          fscanf(f, "%hhu %hhu %hhu\t", &rgb[i*(*w)+j].red, &rgb[i*(*w)+j].green, &rgb[i*(*w)+j].blue);
      }
    }
    fclose(f);
    return rgb;
  } else if (strncmp(header[0], "P6", 2) == 0) {
    fread(rgb, sizeof(struct PPMrgb), (*w) * (*h), f);
    fclose(f);
    return rgb;
  } else {
		fprintf(stderr, "Read Error: Support P3 and P6 only\n");
    fclose(f);
    freeAll(rgb, w, h, d);
    return FAILURE;
  }
}

int writePPMFile(char *filename, struct PPMrgb *rgb, unsigned int *w, unsigned int *h, unsigned int *d, unsigned int m)
{
  int i,j;
  FILE *f;
  f = fopen(filename, "w");
  // write first 4 parameters
  if (m == 3)
    fprintf(f, "P3\n");
  else if (m == 6)
    fprintf(f, "P6\n");
  else {
    fprintf(stderr, "Write Error: Support P3 and P6 only\n");
    freeAll(rgb, w, h, d);
    return FAILURE;
  }
  fprintf(f, "#COM4521 Assignment\n");
  fprintf(f, "%d\n", *w);
  fprintf(f, "%d\n", *h);
  fprintf(f, "%d\n", *d);

  // write P3 and P6 PPM rgb
  if (m == 3) {
    for (i = 0; i < *h; i++){
      for (j = 0; j < *w; j++) {
        if (j == (*w) - 1)
          fprintf(f, "%hhu %hhu %hhu\n", rgb[i*(*w)+j].red, rgb[i*(*w)+j].green, rgb[i*(*w)+j].blue);
        else
          fprintf(f, "%hhu %hhu %hhu\t", rgb[i*(*w)+j].red, rgb[i*(*w)+j].green, rgb[i*(*w)+j].blue);
      }
    }
    fclose(f);
    return SUCCESS;
  } else if (m == 6) {
      fwrite(rgb, sizeof(struct PPMrgb), (*w) * (*h), f);
      fclose(f);
      return SUCCESS;
  } else {
		fprintf(stderr, "Save Error: Support P3 and P6 only\n");
    fclose(f);
    freeAll(rgb, w, h, d);
    return FAILURE;
  }
}

void checkCUDAError(const char  *msg)
{
  hipError_t err  = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

void freeAll(struct PPMrgb *rgb, unsigned int *w, unsigned int *h, unsigned int *d)
{
  free(rgb); free(w); free(h); free(d);
}
